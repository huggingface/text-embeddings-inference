#include "hip/hip_runtime.h"
#include "fmha.h"
#include "fmha_utils.h"

void run_fmha_fwd(Launch_params<FMHA_fprop_params> &launch_params) {
    if (launch_params.params.d <= 32) {
        run_fmha_fwd_hdim32(launch_params);
    } else if (launch_params.params.d <= 64) {
        run_fmha_fwd_hdim64(launch_params);
    } else if (launch_params.params.d <= 128) {
        run_fmha_fwd_hdim128(launch_params);
    }
}

extern "C" void run_mha(
    void *q_ptr,
    void *k_ptr,
    void *v_ptr,
    void *o_ptr,
    void *o_tmp_ptr,
    void *softmax_lse_ptr,

    int32_t *cu_seqlens_q_ptr,
    int32_t *cu_seqlens_k_ptr,

    uint32_t q_row_stride,
    uint32_t k_row_stride,
    uint32_t v_row_stride,
    uint32_t o_row_stride,
    uint32_t o_tmp_row_stride,

    uint32_t q_head_stride,
    uint32_t k_head_stride,
    uint32_t v_head_stride,
    uint32_t o_head_stride,
    uint32_t o_tmp_head_stride,

    uint32_t b,
    uint32_t h,
    uint32_t d,
    float softmax_scale,

    uint32_t seqlen_q,
    uint32_t seqlen_k,

    int is_causal,
    int is_bf16,

    int32_t multi_processor_count,
    int32_t num_splits
) {
    Data_type data_type = !is_bf16 ? DATA_TYPE_FP16 : DATA_TYPE_BF16;

    Launch_params<FMHA_fprop_params> launch_params;

    launch_params.elts_per_thread = 0;
    launch_params.multi_processor_count = multi_processor_count;
    launch_params.stream = 0;
    launch_params.is_dropout = false;
    launch_params.return_softmax = false;

    FMHA_fprop_params &params = launch_params.params;

    // Set the pointers and strides.
    params.q_ptr = q_ptr;
    params.k_ptr = k_ptr;
    params.v_ptr = v_ptr;
    params.o_ptr = o_ptr;
    params.o_tmp_ptr = o_tmp_ptr;

    params.softmax_lse_ptr = softmax_lse_ptr;

    // All stride are in elements, not bytes.

    params.q_row_stride_in_elts = q_row_stride;
    params.k_row_stride_in_elts = k_row_stride;
    params.v_row_stride_in_elts = v_row_stride;
    params.o_row_stride_in_elts = o_row_stride;
    params.o_tmp_row_stride_in_elts = o_tmp_row_stride;

    params.q_head_stride_in_elts = q_head_stride;
    params.k_head_stride_in_elts = k_head_stride;
    params.v_head_stride_in_elts = v_head_stride;
    params.o_head_stride_in_elts = o_head_stride;
    params.o_tmp_head_stride_in_elts = o_tmp_head_stride;

    // Set the dimensions.
    params.h = h;
    params.b = b;
    params.seqlen_q = seqlen_q;
    params.seqlen_k = seqlen_k;
    params.d = d;

    // Set the different scale values.
    const float scale_bmm1 = softmax_scale;
    params.scale_bmm1f = scale_bmm1;
    set_alpha(params.scale_bmm1, scale_bmm1, data_type);

    params.p_dropout = 1.; // probability to keep
    params.p_dropout_in_uint = uint32_t(std::floor(params.p_dropout * 4294967295.0));
    params.p_dropout_in_uint16_t = uint16_t(std::floor(params.p_dropout * 65535.0));
    params.rp_dropout = 1.f / params.p_dropout;
    params.scale_bmm1_rp_dropout = params.rp_dropout * params.scale_bmm1f;
    set_alpha(params.scale_dropout, params.rp_dropout, data_type);

    params.is_bf16 = is_bf16;
    params.is_causal = is_causal;

    params.cu_seqlens_q = cu_seqlens_q_ptr;
    params.cu_seqlens_k = cu_seqlens_k_ptr;

    params.num_splits = num_splits;

    run_fmha_fwd(launch_params);
}
